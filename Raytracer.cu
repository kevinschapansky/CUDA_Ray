#include "hip/hip_runtime.h"
#include "Raytracer.h"

__global__ void CudaTest() {
    
}

Raytracer::Raytracer(int width, int height, std::vector<std::string> rawComponents) {
    Width = width;
    Height = height;
    ParseRawComponents(rawComponents);
}

Image* Raytracer::TraceScene() {
    Image* scene = new Image(Width, Height);
    
    glm::vec3 U(Cam->Right->x, Cam->Right->y, Cam->Right->z);
    glm::vec3 V(Cam->Up->x, Cam->Up->y, Cam->Up->z);
    glm::vec3 W(1.0);
    float left = -1 * glm::length(*Cam->Right) / 2.0;
    float right = -left;
    float top = glm::length(*Cam->Up) / 2.0;
    float bottom = -top;
    
    U = glm::normalize(U);
    V = glm::normalize(V);
    W = glm::normalize(glm::cross(U, V));
    
    for (int i = 0; i < Width; i++) {
        for (int j = 0; j < Height; j++) {
            float Us = left + (right - left) * ((i + 0.5) / ((float) Width));
            float Vs = bottom + (top - bottom) * ((j + 0.5) / ((float) Height));
            glm::vec3 sPrime = *Cam->Location + Us * U + Vs * V + 1.0f * W;
            glm::vec3 d = glm::normalize(*Cam->Location - sPrime);
            glm::vec3 p0 = *Cam->Location;
            color_t color;
            float closestIntersection = FLT_MAX;
            float curIntersection;
            
            color.r = 0;
            color.g = 0;
            color.b = 0;
            color.f = 0;
            
            for (int k = 0; k < Spheres.size(); k++) {
                float A = glm::dot(d, d);
                float B = 2 * glm::dot(d, (p0 - *Spheres[k]->Position));
                float C = glm::dot((p0 - *Spheres[k]->Position), (p0 - *Spheres[k]->Position))
                - Spheres[k]->Radius * Spheres[k]->Radius;
                float descriminant = B * B - 4 * A * C;
                glm::vec3 colorVec = *Spheres[k]->Pig->Color;
                
                if (descriminant < 0) {
                    continue;
                } else if (descriminant == 0) {
                    curIntersection = -B / (2 * A);
                    
                    if (curIntersection < closestIntersection) {
                        closestIntersection = curIntersection;
                        color.r = colorVec.x;
                        color.g = colorVec.y;
                        color.b = colorVec.z;
                        color.f = Spheres[k]->Pig->Filter;
                    }
                } else {
                    float root = std::sqrt(descriminant);
                    curIntersection = min(((-B + root) / (2 * A)), ((-B - root) / (2 * A)));
                    
                    if (curIntersection < closestIntersection) {
                        closestIntersection = curIntersection;
                        color.r = colorVec.x;
                        color.g = colorVec.y;
                        color.b = colorVec.z;
                        color.f = Spheres[k]->Pig->Filter;
                    }
                }
            }
            for (int k = 0; k < Planes.size(); k++) {
                glm::vec3 P = glm::normalize(*Planes[k]->Normal) * -Planes[k]->Distance;
                float denom = glm::dot(d, *Planes[k]->Normal);
                float t = glm::dot((P - p0), *Planes[k]->Normal) / denom;
                glm::vec3 colorVec = *Planes[k]->Pig->Color;
                
                if (denom < 0.01) {
                    continue;
                }
                
                if (t < closestIntersection) {
                    closestIntersection = t;
                    color.r = colorVec.x;
                    color.g = colorVec.y;
                    color.b = colorVec.z;
                    color.f = Planes[k]->Pig->Filter;
                }
            }
            scene->pixel(i, j, color);
        }
    }
    
    return scene;
}

void Raytracer::GenerateRays() {
    glm::vec3 U(Cam->Right->x, Cam->Right->y, Cam->Right->z);
    glm::vec3 V(Cam->Up->x, Cam->Up->y, Cam->Up->z);
    glm::vec3 W(1.0);
    float left = - ((float) Width) / ((float) Height);
    float right = -left;
    float top = 1;
    float bottom = -1;
    
    U = glm::normalize(U);
    V = glm::normalize(V);
    W = glm::normalize(glm::cross(U, V));
    
    for (int i = 0; i < Width; i++) {
        for (int j = 0; j < Height; j++) {
            float Us = left + (right - left) * ((i + 0.5) / ((float) Width));
            float Vs = bottom + (top - bottom) * ((j + 0.5) / ((float) Height));
            glm::vec3 sPrime = *Cam->Location + Us * U + Vs * V + -1.0f * W;
            glm::vec3 d = glm::normalize(*Cam->Location - sPrime);
            glm::vec3 p0 = *Cam->Location;
            glm::vec3 color(0);
            float closestIntersection = FLT_MAX;
            float curIntersection;
            
            for (int k = 0; k < Spheres.size(); k++) {
                float A = glm::dot(d, d);
                float B = 2 * glm::dot(d, (p0 - *Spheres[k]->Position));
                float C = glm::dot((p0 - *Spheres[k]->Position), (p0 - *Spheres[k]->Position))
                - Spheres[k]->Radius * Spheres[k]->Radius;
                float descriminant = B * B - 4 * A * C;
                
                if (descriminant < 0) {
                    continue;
                } else if (descriminant == 0) {
                    curIntersection = -B / (2 * A);
                    
                    if (curIntersection < closestIntersection) {
                        closestIntersection = curIntersection;
                        color = *Spheres[k]->Pig->Color;
                    }
                } else {
                    float root = std::sqrt(descriminant);
                    curIntersection = min(((-B + root) / (2 * A)), ((-B - root) / (2 * A)));
                    
                    if (curIntersection < closestIntersection) {
                        closestIntersection = curIntersection;
                        color = *Spheres[k]->Pig->Color;
                    }
                }
            }
            
        }
    }
}

void Raytracer::ParseRawComponents(std::vector<std::string> components) {
    for (int i = 0; i < components.size(); i++) {
        std::string curComp = components[i];
        
        if (std::string::npos != curComp.find("camera")) {
            Cam = new Camera(curComp);
        } else if (std::string::npos != curComp.find("light_source")) {
            Lights.push_back(new LightSource(curComp));
        } else if (std::string::npos != curComp.find("sphere")) {
            Spheres.push_back(new Sphere(curComp));
        } else if (std::string::npos != curComp.find("box")) {
            Boxes.push_back(new Box(curComp));
        } else if (std::string::npos != curComp.find("cone")) {
           Cones.push_back(new Cone(curComp));
        } else if (std::string::npos != curComp.find("plane")) {
            Planes.push_back(new Plane(curComp));
        } else if (std::string::npos != curComp.find("triangle")) {
            Triangles.push_back(new Triangle(curComp));
        }
    }
}