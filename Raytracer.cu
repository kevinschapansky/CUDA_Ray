#include "hip/hip_runtime.h"
#include "Raytracer.h"

__device__ Intersection GetIntersection(SceneData data, Ray ray, float minInt, float maxInt) {
    Intersection closestInt;
    glm::vec3 d_model;
    glm::vec3 p0_model;
    
    closestInt.T = maxInt;
    closestInt.ClosestShape = -1;
    
    for (int i = 0; i < data.NumShapes; i++) {
        d_model = glm::vec3(data.Shapes[i].InverseTransform * glm::vec4(ray.D, 0));
        p0_model = glm::vec3(data.Shapes[i].InverseTransform * glm::vec4(ray.P0, 1));
        
        if (data.Shapes[i].Type == Shape::SPHERE) {
            float A = glm::dot(d_model, d_model);
            float B = 2.0f * glm::dot(d_model, (p0_model - data.Shapes[i].Position));
            float C = glm::dot((p0_model - data.Shapes[i].Position), (p0_model - data.Shapes[i].Position))
            - data.Shapes[i].Radius * data.Shapes[i].Radius;
            float descriminant = B * B - 4 * A * C;
            float t;
            
            if (descriminant < 0) {
                continue;
            } else if (descriminant == 0) {
                t = -B / (2.0f * A);
                
                if (t < closestInt.T && t > minInt && t < maxInt) {
                    closestInt.T = t;
                    closestInt.ClosestShape = i;
                    closestInt.SurfaceNormal = glm::vec3(glm::inverseTranspose(data.Shapes[i].Transform) * glm::vec4((p0_model + t * d_model) - data.Shapes[i].Position, 0));
                }
            } else {
                float root = std::sqrt(descriminant);
                t = min(((-B + root) / (2.0f * A)), ((-B - root) / (2.0f * A)));
                if (t < closestInt.T && t > minInt && t < maxInt) {
                    closestInt.T = t;
                    closestInt.ClosestShape = i;
                    closestInt.SurfaceNormal = glm::vec3(glm::inverseTranspose(data.Shapes[i].Transform) * glm::vec4((p0_model + t * d_model) - data.Shapes[i].Position, 0));
                }
            }
        } else if (data.Shapes[i].Type == Shape::PLANE) {
            glm::vec3 P = glm::normalize(data.Shapes[i].Normal) * data.Shapes[i].Distance;
            float denom = glm::dot(d_model, data.Shapes[i].Normal);
            float t = glm::dot((P - p0_model), data.Shapes[i].Normal) / denom;

            if (abs(denom) > 0.001f && t < closestInt.T && t > minInt && t < maxInt) {
                closestInt.T = t;
                closestInt.ClosestShape = i;
                closestInt.SurfaceNormal = glm::vec3(glm::inverseTranspose(data.Shapes[i].Transform) * glm::vec4(data.Shapes[i].Normal, 0));
            }
        }
    }
    return closestInt;
}

__device__ glm::vec3 GetLightingAtIntersection(SceneData data, Intersection inter, Ray ray) {
    glm::vec3 intersectionColor(0);
    Shape curShape = data.Shapes[inter.ClosestShape];
    Intersection lightInt;
    glm::vec3 intersectionPoint = ray.P0 + inter.T * ray.D;
    glm::vec3 lightPosition_model;
    Ray lightRay;
    
    lightRay.P0 = intersectionPoint;
    intersectionColor = curShape.Pig.Color * curShape.Fin.Ambient;
    
    for (int i = 0; i < data.NumLights; i++) {
        lightRay.D = glm::normalize(data.Lights[i].Position - intersectionPoint);
        lightInt = GetIntersection(data, lightRay, 0.01f,
                                   glm::length(data.Lights[i].Position - intersectionPoint));
        
        if (lightInt.ClosestShape < 0) {
            glm::vec3 reflection = - 2.0f * (max(glm::dot(lightRay.D, inter.SurfaceNormal), 0.0f)) * inter.SurfaceNormal + lightRay.D;
            
            intersectionColor += curShape.Pig.Color * max(glm::dot(inter.SurfaceNormal, lightRay.D), 0.0f) * data.Lights[i].Color * curShape.Fin.Diffuse;
            intersectionColor += curShape.Pig.Color * pow(max(glm::dot(-ray.D, reflection), 0.0f), 1.0f / curShape.Fin.Roughness) * data.Lights[i].Color * curShape.Fin.Specular;
        }
    }
    intersectionColor.x = min(intersectionColor.x, 1.0f);
    intersectionColor.y = min(intersectionColor.y, 1.0f);
    intersectionColor.z = min(intersectionColor.z, 1.0f);
    
    return intersectionColor;
}

__global__ void CUDATrace(SceneData data, color_t *scenePixels, int N) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int sceneIx = i * data.Height + j;
    
    if (i * blockDim.x + j > N) return;
    scenePixels[sceneIx].r = 0;
    scenePixels[sceneIx].g = 0;
    scenePixels[sceneIx].b = 0;
    scenePixels[sceneIx].f = 0;
    
    
    float Us = data.Params.Left + (data.Params.Right - data.Params.Left) * ((i + 0.5f) / ((float) data.Width));
    float Vs = data.Params.Bottom + (data.Params.Top - data.Params.Bottom) * ((j + 0.5f) / ((float) data.Height));
    glm::vec3 sPrime = data.Cam.Location + Us * data.Params.U + Vs * data.Params.V + -1.0f * data.Params.W;
    Ray castRay;
    glm::vec3 pixelColor;
    
    castRay.D = glm::normalize(sPrime - data.Cam.Location);
    castRay.P0 = data.Cam.Location;
    
    Intersection closestInt = GetIntersection(data, castRay, FLT_MIN, FLT_MAX);
    
    if (closestInt.ClosestShape >= 0) {
        glm::vec3 pixelColor = GetLightingAtIntersection(data, closestInt, castRay);
        scenePixels[sceneIx].r = pixelColor.x;
        scenePixels[sceneIx].g = pixelColor.y;
        scenePixels[sceneIx].b = pixelColor.z;
        scenePixels[sceneIx].f = 0;
    }
     
}



Raytracer::Raytracer(int width, int height, std::vector<std::string> rawComponents) {
    Data.Width = width;
    Data.Height = height;
    
    ParseRawComponents(rawComponents);
}

Image* Raytracer::TraceScene() {
    TracedScene = new Image(Data.Width, Data.Height);
    
    Data.Params.U = glm::normalize(glm::vec3(Data.Cam.Right.x, Data.Cam.Right.y, Data.Cam.Right.z));
    Data.Params.V = glm::normalize(glm::vec3(Data.Cam.Up.x, Data.Cam.Up.y, Data.Cam.Up.z));
    Data.Params.W = glm::normalize(glm::cross(Data.Params.U, Data.Params.V));
    
    Data.Params.Left = -1 * glm::length(Data.Cam.Right) / 2.0;
    Data.Params.Right = -Data.Params.Left;
    Data.Params.Top = glm::length(Data.Cam.Up) / 2.0;
    Data.Params.Bottom = -Data.Params.Top;
    
    SetupAndLaunchCUDA();
    
    return TracedScene;
}

void Raytracer::SetupAndLaunchCUDA() {
    dim3 threadsPerBlock(8,8);
    dim3 numBlocks(Data.Width / threadsPerBlock.x,
                   Data.Height / threadsPerBlock.y);
    
    SceneData cudaData_d = Data;
    
    color_t *scenePixels_h = new color_t[Data.Width * Data.Height];
    color_t *scenePixels_d;
    
    LightSource *lights_d;
    Shape *shapes_d;
    
    HandleCUDAError(hipMalloc((void **) &lights_d, Data.NumLights * sizeof(LightSource)));
    HandleCUDAError(hipMalloc((void **) &shapes_d, Data.NumShapes * sizeof(Shape)));
    
    HandleCUDAError(hipMemcpy(lights_d, Data.Lights, Data.NumLights * sizeof(LightSource), hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(shapes_d, Data.Shapes, Data.NumShapes * sizeof(Shape), hipMemcpyHostToDevice));
    
    HandleCUDAError(hipMalloc((void **) &scenePixels_d, Data.Width * Data.Height * sizeof(color_t)));
               
    cudaData_d.Lights = lights_d;
    cudaData_d.Shapes = shapes_d;
    
    CUDATrace <<<numBlocks, threadsPerBlock>>> (cudaData_d, scenePixels_d, Data.Width * Data.Height);
    
    HandleCUDAError(hipMemcpy(scenePixels_h, scenePixels_d, Data.Width * Data.Height * sizeof(color_t), hipMemcpyDeviceToHost));
    
    for (int i = 0; i < Data.Width; i++) {
        for (int j = 0; j < Data.Height; j++) {
            TracedScene->pixel(i, j, scenePixels_h[i * Data.Height + j]);
        }
    }
    HandleCUDAError(hipFree(lights_d));
    HandleCUDAError(hipFree(shapes_d));
    HandleCUDAError(hipFree(scenePixels_d));
}

void Raytracer::ParseRawComponents(std::vector<std::string> components) {
    for (int i = 0; i < components.size(); i++) {
        std::string curComp = components[i];
        
        if (std::string::npos != curComp.find("camera")) {
            Data.Cam = *(new Camera(curComp));
        } else if (std::string::npos != curComp.find("light_source")) {
            Lights.push_back(new LightSource(curComp));
        } else if (std::string::npos != curComp.find("sphere")) {
            Shapes.push_back(new Shape(curComp, Shape::SPHERE));
        } else if (std::string::npos != curComp.find("plane")) {
            Shape *curShape = new Shape(curComp, Shape::PLANE);
            Shapes.push_back(curShape);
            
            //printf("Normal: %f, %f, %f Distance: %f\n", curShape->Normal.x, curShape->Normal.y, curShape->Normal.z, curShape->Distance);
        } 
    }
    Data.Lights = new LightSource[Lights.size()];
    Data.NumLights = Lights.size();
    
    for (int i = 0; i < Lights.size(); i++) {
        Data.Lights[i] = *Lights[i];
    }
    
    Data.Shapes = new Shape[Shapes.size()];
    Data.NumShapes = Shapes.size();
    
    for (int i = 0; i < Shapes.size(); i++) {
        Data.Shapes[i] = *Shapes[i];
    }
}

void Raytracer::HandleCUDAError(hipError_t error) {
    if (error != hipSuccess) {
        printf("CUDA Problem: %s\n", hipGetErrorString(error));
        exit(-1);
    }
}
