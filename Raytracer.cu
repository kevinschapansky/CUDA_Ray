#include "hip/hip_runtime.h"
#include "Raytracer.h"

__device__ Intersection GetIntersection(SceneData data, Ray ray, float minInt, float maxInt) {
    Intersection closestInt;
    glm::vec3 d_model;
    glm::vec3 p0_model;
    
    closestInt.T = maxInt;
    closestInt.ClosestShape = -1;
    
    for (int i = 0; i < data.NumShapes; i++) {
        d_model = glm::vec3(glm::inverse(data.Shapes[i].Transform) * glm::vec4(ray.D, 0));
        p0_model = glm::vec3(glm::inverse(data.Shapes[i].Transform) * glm::vec4(ray.P0, 1));
        
        if (data.Shapes[i].Type == Shape::SPHERE) {
            float A = glm::dot(d_model, d_model);
            float B = 2.0f * glm::dot(d_model, (p0_model - data.Shapes[i].Position));
            float C = glm::dot((p0_model - data.Shapes[i].Position), (p0_model - data.Shapes[i].Position))
            - data.Shapes[i].Radius * data.Shapes[i].Radius;
            float descriminant = B * B - 4 * A * C;
            float t;
            
            if (descriminant < 0) {
                continue;
            } else if (descriminant == 0) {
                t = -B / (2.0f * A);
                
                if (t < closestInt.T && t > minInt && t < maxInt) {
                    closestInt.T = t;
                    closestInt.ClosestShape = i;
                }
            } else {
                float root = std::sqrt(descriminant);
                t = min(((-B + root) / (2.0f * A)), ((-B - root) / (2.0f * A)));
                if (t < closestInt.T && t > minInt && t < maxInt) {
                    closestInt.T = t;
                    closestInt.ClosestShape = i;
                }
            }
        } else if (data.Shapes[i].Type == Shape::PLANE) {
            glm::vec3 P = glm::normalize(data.Shapes[i].Normal) * -data.Shapes[i].Distance;
            float denom = glm::dot(d_model, data.Shapes[i].Normal);
            float t = glm::dot((P - p0_model), data.Shapes[i].Normal) / denom;

            if (denom > 0.001f && t < closestInt.T && t > minInt && t < maxInt) {
                closestInt.T = t;
                closestInt.ClosestShape = i;
            }
        }
    }
    return closestInt;
}

__global__ void CUDATrace(SceneData data, color_t *scenePixels, int N) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int sceneIx = i * data.Height + j;
    
    if (i * blockDim.x + j > N) return;
    scenePixels[sceneIx].r = 0;
    scenePixels[sceneIx].g = 0;
    scenePixels[sceneIx].b = 0;
    scenePixels[sceneIx].f = 0;
    
    
    float Us = data.Params.Left + (data.Params.Right - data.Params.Left) * ((i + 0.5f) / ((float) data.Width));
    float Vs = data.Params.Bottom + (data.Params.Top - data.Params.Bottom) * ((j + 0.5f) / ((float) data.Height));
    glm::vec3 sPrime = data.Cam.Location + Us * data.Params.U + Vs * data.Params.V + -1.0f * data.Params.W;
    Ray castRay;
    glm::vec3 pixelColor;
    
    castRay.D = glm::normalize(data.Cam.Location - sPrime);
    castRay.P0 = data.Cam.Location;
    
    Intersection closestInt = GetIntersection(data, castRay, -FLT_MAX, FLT_MAX);
    
    if (closestInt.ClosestShape >= 0) {
        pixelColor = data.Shapes[closestInt.ClosestShape].Pig.Color;
        scenePixels[sceneIx].r = pixelColor.x;
        scenePixels[sceneIx].g = pixelColor.y;
        scenePixels[sceneIx].b = pixelColor.z;
        scenePixels[sceneIx].f = data.Shapes[closestInt.ClosestShape].Pig.Filter;
    }
     
}



Raytracer::Raytracer(int width, int height, std::vector<std::string> rawComponents) {
    Data.Width = width;
    Data.Height = height;
    
    ParseRawComponents(rawComponents);
}

Image* Raytracer::TraceScene() {
    TracedScene = new Image(Data.Width, Data.Height);
    
    Data.Params.U = glm::normalize(glm::vec3(Data.Cam.Right.x, Data.Cam.Right.y, Data.Cam.Right.z));
    Data.Params.V = glm::normalize(glm::vec3(Data.Cam.Up.x, Data.Cam.Up.y, Data.Cam.Up.z));
    Data.Params.W = glm::normalize(glm::cross(Data.Params.U, Data.Params.V));
    
    Data.Params.Left = -1 * glm::length(Data.Cam.Right) / 2.0;
    Data.Params.Right = -Data.Params.Left;
    Data.Params.Top = glm::length(Data.Cam.Up) / 2.0;
    Data.Params.Bottom = -Data.Params.Top;
    
    SetupAndLaunchCUDA();
    
    return TracedScene;
}

void Raytracer::SetupAndLaunchCUDA() {
    dim3 threadsPerBlock(8,8);
    dim3 numBlocks(Data.Width / threadsPerBlock.x,
                   Data.Height / threadsPerBlock.y);
    
    SceneData cudaData_d = Data;
    
    color_t *scenePixels_h = new color_t[Data.Width * Data.Height];
    color_t *scenePixels_d;
    
    LightSource *lights_d;
    Shape *shapes_d;
    
    HandleCUDAError(hipMalloc((void **) &lights_d, Data.NumLights * sizeof(LightSource)));
    HandleCUDAError(hipMalloc((void **) &shapes_d, Data.NumShapes * sizeof(Shape)));
    
    HandleCUDAError(hipMemcpy(lights_d, Data.Lights, Data.NumLights * sizeof(LightSource), hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(shapes_d, Data.Shapes, Data.NumShapes * sizeof(Shape), hipMemcpyHostToDevice));
    
    HandleCUDAError(hipMalloc((void **) &scenePixels_d, Data.Width * Data.Height * sizeof(color_t)));
               
    cudaData_d.Lights = lights_d;
    cudaData_d.Shapes = shapes_d;
    
    CUDATrace <<<numBlocks, threadsPerBlock>>> (cudaData_d, scenePixels_d, Data.Width * Data.Height);
    
    HandleCUDAError(hipMemcpy(scenePixels_h, scenePixels_d, Data.Width * Data.Height * sizeof(color_t), hipMemcpyDeviceToHost));
    
    for (int i = 0; i < Data.Width; i++) {
        for (int j = 0; j < Data.Height; j++) {
            TracedScene->pixel(i, j, scenePixels_h[i * Data.Height + j]);
        }
    }
    HandleCUDAError(hipFree(lights_d));
    HandleCUDAError(hipFree(shapes_d));
    HandleCUDAError(hipFree(scenePixels_d));
}

void Raytracer::ParseRawComponents(std::vector<std::string> components) {
    for (int i = 0; i < components.size(); i++) {
        std::string curComp = components[i];
        
        if (std::string::npos != curComp.find("camera")) {
            Data.Cam = *(new Camera(curComp));
        } else if (std::string::npos != curComp.find("light_source")) {
            Lights.push_back(new LightSource(curComp));
        } else if (std::string::npos != curComp.find("sphere")) {
            Shapes.push_back(new Shape(curComp, Shape::SPHERE));
        } else if (std::string::npos != curComp.find("plane")) {
            Shape *curShape = new Shape(curComp, Shape::PLANE);
            Shapes.push_back(curShape);
            
            //printf("Normal: %f, %f, %f Distance: %f\n", curShape->Normal.x, curShape->Normal.y, curShape->Normal.z, curShape->Distance);
        } 
    }
    Data.Lights = new LightSource[Lights.size()];
    Data.NumLights = Lights.size();
    
    for (int i = 0; i < Lights.size(); i++) {
        Data.Lights[i] = *Lights[i];
    }
    
    Data.Shapes = new Shape[Shapes.size()];
    Data.NumShapes = Shapes.size();
    
    for (int i = 0; i < Shapes.size(); i++) {
        Data.Shapes[i] = *Shapes[i];
    }
}

void Raytracer::HandleCUDAError(hipError_t error) {
    if (error != hipSuccess) {
        printf("CUDA Problem: %s\n", hipGetErrorString(error));
        exit(-1);
    }
}
