#include "hip/hip_runtime.h"
#include "Raytracer.h"

__global__ void CUDATrace(SceneData data, color_t *scenePixels, int N) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    if (i * blockDim.x + j > N) return;

    float Us = data.Params.Left + (data.Params.Right - data.Params.Left) * ((i + 0.5f) / ((float) data.Width));
    float Vs = data.Params.Bottom + (data.Params.Top - data.Params.Bottom) * ((j + 0.5f) / ((float) data.Height));
    glm::vec3 sPrime = data.Cam.Location + Us * data.Params.U + Vs * data.Params.V + -1.0f * data.Params.W;
    glm::vec3 d = glm::normalize(data.Cam.Location - sPrime);
    glm::vec3 p0 = data.Cam.Location;
    color_t color;
    float closestIntersection = FLT_MAX;
    float curIntersection;
    
    for (int k = 0; k < data.NumSpheres; k++) {
        glm::vec3 d_model = glm::vec3(glm::inverse(data.Spheres[k].Transform) * glm::vec4(d, 0));
        glm::vec3 p0_model = glm::vec3(glm::inverse(data.Spheres[k].Transform) * glm::vec4(p0, 1));
        float A = glm::dot(d_model, d_model);
        float B = 2.0f * glm::dot(d_model, (p0_model - data.Spheres[k].Position));
        float C = glm::dot((p0_model - data.Spheres[k].Position), (p0_model - data.Spheres[k].Position))
        - data.Spheres[k].Radius * data.Spheres[k].Radius;
        float descriminant = B * B - 4 * A * C;
        glm::vec3 colorVec = data.Spheres[k].Pig.Color;
        
        if (descriminant < 0) {
            continue;
        } else if (descriminant == 0) {
            curIntersection = -B / (2.0f * A);
            
            if (curIntersection < closestIntersection) {
                closestIntersection = curIntersection;
                color.r = colorVec.x;
                color.g = colorVec.y;
                color.b = colorVec.z;
                color.f = data.Spheres[k].Pig.Filter;
            }
        } else {
            float root = std::sqrt(descriminant);
            curIntersection = min(((-B + root) / (2.0f * A)), ((-B - root) / (2.0f * A)));
            
            if (curIntersection < closestIntersection) {
                closestIntersection = curIntersection;
                color.r = colorVec.x;
                color.g = colorVec.y;
                color.b = colorVec.z;
                color.f = data.Spheres[k].Pig.Filter;
            }
        }
    }
    for (int k = 0; k < data.NumPlanes; k++) {
        glm::vec3 d_model = glm::vec3(glm::inverse(data.Planes[k].Transform) * glm::vec4(d, 0));
        glm::vec3 p0_model = glm::vec3(glm::inverse(data.Planes[k].Transform) * glm::vec4(p0, 1));
        glm::vec3 P = glm::normalize(data.Planes[k].Normal) * -data.Planes[k].Distance;
        float denom = glm::dot(d_model, data.Planes[k].Normal);
        float t = glm::dot((P - p0_model), data.Planes[k].Normal) / denom;
        glm::vec3 colorVec = data.Planes[k].Pig.Color;
        
        if (denom < 0.001f) {
            continue;
        }
        
        if (t < closestIntersection) {
            closestIntersection = t;
            color.r = colorVec.x;
            color.g = colorVec.y;
            color.b = colorVec.z;
            color.f = data.Planes[k].Pig.Filter;
        }
    }
     
    scenePixels[i * data.Height + j] = color;
}

Raytracer::Raytracer(int width, int height, std::vector<std::string> rawComponents) {
    Data.Width = width;
    Data.Height = height;
    
    ParseRawComponents(rawComponents);
}

Image* Raytracer::TraceScene() {
    TracedScene = new Image(Data.Width, Data.Height);
    
    Data.Params.U = glm::normalize(glm::vec3(Data.Cam.Right.x, Data.Cam.Right.y, Data.Cam.Right.z));
    Data.Params.V = glm::normalize(glm::vec3(Data.Cam.Up.x, Data.Cam.Up.y, Data.Cam.Up.z));
    Data.Params.W = glm::normalize(glm::cross(Data.Params.U, Data.Params.V));
    
    Data.Params.Left = -1 * glm::length(Data.Cam.Right) / 2.0;
    Data.Params.Right = -Data.Params.Left;
    Data.Params.Top = glm::length(Data.Cam.Up) / 2.0;
    Data.Params.Bottom = -Data.Params.Top;
    
    SetupAndLaunchCUDA();
    
    return TracedScene;
}

void Raytracer::SetupAndLaunchCUDA() {
    dim3 threadsPerBlock(8,8);
    dim3 numBlocks(Data.Width / threadsPerBlock.x,
                   Data.Height / threadsPerBlock.y);
    
    SceneData cudaData_d = Data;
    color_t *scenePixels_h = new color_t[Data.Width * Data.Height];
    color_t *scenePixels_d;
    
    LightSource *lights_d;
    Sphere *spheres_d;
    Plane *planes_d;
    
    HandleCUDAError(hipMalloc((void **) &lights_d, Data.NumLights * sizeof(LightSource)));
    HandleCUDAError(hipMalloc((void **) &spheres_d, Data.Width * Data.NumSpheres * sizeof(Sphere)));
    HandleCUDAError(hipMalloc((void **) &planes_d, Data.Width * Data.NumPlanes * sizeof(Plane)));
    
    HandleCUDAError(hipMemcpy(lights_d, Data.Lights, Data.NumLights * sizeof(LightSource), hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(spheres_d, Data.Spheres, Data.NumSpheres * sizeof(Sphere), hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(planes_d, Data.Planes, Data.NumPlanes * sizeof(Plane), hipMemcpyHostToDevice));
    
    HandleCUDAError(hipMalloc((void **) &scenePixels_d, Data.Width * Data.Height * sizeof(color_t)));
               
    cudaData_d.Lights = lights_d;
    cudaData_d.Spheres = spheres_d;
    cudaData_d.Planes = planes_d;
    
    CUDATrace <<<numBlocks, threadsPerBlock>>> (cudaData_d, scenePixels_d, Data.Width * Data.Height);
    
    HandleCUDAError(hipMemcpy(scenePixels_h, scenePixels_d, Data.Width * Data.Height * sizeof(color_t), hipMemcpyDeviceToHost));
    
    for (int i = 0; i < Data.Width; i++) {
        for (int j = 0; j < Data.Height; j++) {
            TracedScene->pixel(i, j, scenePixels_h[i * Data.Height + j]);
        }
    }
}

void Raytracer::ParseRawComponents(std::vector<std::string> components) {
    for (int i = 0; i < components.size(); i++) {
        std::string curComp = components[i];
        
        if (std::string::npos != curComp.find("camera")) {
            Data.Cam = Camera(curComp);
        } else if (std::string::npos != curComp.find("light_source")) {
            Lights.push_back(LightSource(curComp));
        } else if (std::string::npos != curComp.find("sphere")) {
            Spheres.push_back(Sphere(curComp));
        } else if (std::string::npos != curComp.find("plane")) {
            Planes.push_back(Plane(curComp));
        } else if (std::string::npos != curComp.find("triangle")) {
            Triangles.push_back(Triangle(curComp));
        }
    }
    Data.Lights = Lights.data();
    Data.NumLights = Lights.size();
    
    Data.Spheres = Spheres.data();
    Data.NumSpheres = Spheres.size();
    
    Data.Planes = Planes.data();
    Data.NumPlanes = Planes.size();
}

void Raytracer::HandleCUDAError(hipError_t error) {
    if (error != hipSuccess) {
        printf("CUDA Problem: %s\n", hipGetErrorString(error));
        exit(-1);
    }
}
