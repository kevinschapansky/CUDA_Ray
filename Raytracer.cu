#include "hip/hip_runtime.h"
#include "Raytracer.h"

__global__ void CudaTest() {
    
}

Raytracer::Raytracer(std::vector<std::string> rawComponents) {
    ParseRawComponents(rawComponents);
}

void Raytracer::ParseRawComponents(std::vector<std::string> components) {
    for (int i = 0; i < components.size(); i++) {
        std::string curComp = components[i];
        
        if (std::string::npos != curComp.find("camera")) {
            Cam = new Camera(curComp);
        } else if (std::string::npos != curComp.find("light_source")) {
            Lights.push_back(new LightSource(curComp));
        } else if (std::string::npos != curComp.find("sphere")) {
            Spheres.push_back(new Sphere(curComp));
        } else if (std::string::npos != curComp.find("box")) {
            Boxes.push_back(new Box(curComp));
        } else if (std::string::npos != curComp.find("cone")) {
           Cones.push_back(new Cone(curComp));
        } else if (std::string::npos != curComp.find("plane")) {
            Planes.push_back(new Plane(curComp));
        } else if (std::string::npos != curComp.find("triangle")) {
            Triangles.push_back(new Triangle(curComp));
        }
    }
}